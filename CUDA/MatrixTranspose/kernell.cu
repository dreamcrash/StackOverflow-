#include "hip/hip_runtime.h"
/** Original Question : https://stackoverflow.com/questions/13782012

How to transpose a matrix in CUDA/cublas?

Say I have a matrix with a dimension of A*B on GPU, where B (number of columns) 
is the leading dimension assuming a C style. Is there any method in CUDA (or cublas) 
to transpose this matrix to FORTRAN style, where A (number of rows) becomes the leading dimension?

It is even better if it could be transposed during host->device transfer while keep the original data unchanged.



**/


__global__ void transposeNaive(float *odata, float* idata,
int width, int height, int nreps)
{
    int xIndex = blockIdx.x*TILE_DIM + threadIdx.x;
    int yIndex = blockIdx.y*TILE_DIM + threadIdx.y;
    int index_in = xIndex + width * yIndex;
    int index_out = yIndex + height * xIndex;

    for (int r=0; r < nreps; r++)
    {
        for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS)
        {
          odata[index_out+i] = idata[index_in+i*width];
        }
    }
}

