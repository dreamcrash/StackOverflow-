#include "hip/hip_runtime.h"
/** Original Question : https://stackoverflow.com/questions/13160617

I have two arrays, a and b, and I would like to compute the "min convolution" to produce result c. Simple pseudo code looks like the following:

for i = 0 to size(a)+size(b)
    c[i] = inf
    for j = 0 to size(a)
        if (i - j >= 0) and (i - j < size(b))
            c[i] = min(c[i], a[j] + b[i-j])
(edit: changed loops to start at 0 instead of 1)

If the min were instead a sum, we could use a Fast Fourier Transform (FFT), but in the min case, there is no such analog. Instead, I'd like to make this simple algorithm as fast as possible by using a GPU (CUDA). I'd be happy to find existing code that does this (or code that implements the sum case without FFTs, so that I could adapt it for my purposes), but my search so far hasn't turned up any good results. My use case will involve a's and b's that are of size between 1,000 and 100,000.

Questions:

Does code to do this efficiently already exist?
If I am going to implement this myself, structurally, how should the CUDA kernel look so as to maximize efficiency? I've tried a simple solution where each c[i] is computed by a separate thread, but this doesn't seem like the best way. Any tips in terms of how to set up thread block structure and memory access patterns?

**/


/** A faster version **/
__global__ void convAgB(double *a, double *b, double *c, int sa, int sb)
{
    int i = (threadIdx.x + blockIdx.x * blockDim.x);
    int idT = threadIdx.x;
    int out,j;

    __shared__ double c_local [512];

    c_local[idT] = c[i];

    out = (i > sa) ? sa : i + 1;
    j   = (i > sb) ? i - sb + 1 : 1;

    for(; j < out; j++)
    {    
       if(c_local[idT] > a[j] + b[i-j])
          c_local[idT] = a[j] + b[i-j]; 
    }   

    c[i] = c_local[idT];
} 

/** Older Version **/
__global__ void convAgB(double *a, double *b, double *c, int sa, int sb)
{
    int size = sa+sb;

    int idT = (threadIdx.x + blockIdx.x * blockDim.x);
    int out,j;


    for(int i = idT; i < size; i += blockDim.x * gridDim.x)
    {
        if(i > sa) out = sa;
        else out = i + 1;

        if(i > sb) j = i - sb + 1;
        else j = 1;


        for(; j < out; j++)
        {
                if(c[i] > a[j] + b[i-j])
                    c[i] = a[j] + b[i-j];
        }
    }
}

