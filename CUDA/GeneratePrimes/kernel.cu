
#include <hip/hip_runtime.h>
/*** Original Question : https://stackoverflow.com/questions/13215614/

I am new to CUDA. I am trying to parallelize the following code. Right now it's sitting on kernel but is not using threads at all, thus slow. I tried to use this answer but to no avail so far.

The kernel is supposed to generate first n prime numbers, put them into device_primes array and this array is later accessed from host. The code is correct and works fine in serial version but I need to speed it up, perhaps with use of shared memory.

//CUDA kernel code
__global__ void generatePrimes(int* device_primes, int n) 
{
//int i = blockIdx.x * blockDim.x + threadIdx.x;
//int j = blockIdx.y * blockDim.y + threadIdx.y;

int counter = 0;
int c = 0;

for (int num = 2; counter < n; num++)
{       
    for (c = 2; c <= num - 1; c++)
    { 
        if (num % c == 0) //not prime
        {
            break;
        }
    }
    if (c == num) //prime
    {
        device_primes[counter] = num;
        counter++;
    }
}
}
My current, preliminary, and definitely wrong attempt to parallelize this looks like the following:

//CUDA kernel code
__global__ void generatePrimes(int* device_primes, int n) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int num = i + 2; 
    int c = j + 2;
    int counter = 0;

    if ((counter >= n) || (c > num - 1))
    {
        return;
    }
    if (num % c == 0) //not prime
    {
    
    }
    if (c == num) //prime
    {
       device_primes[counter] = num;
       counter++;
    }
    num++;
    c++;
}
But this code populates the array with data that does not make sense. In addition, many values are zeroes. Thanks in advance for any help, it's appreciated.

***/


__global__ void getPrimes(int *device_primes,int n)
{ 
    int c = 0;
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int num = thread_id;

    if (thread_id == 0) device_primes[0] = 1;
    __syncthreads();

    while(device_primes[0] < n)
    {

        for (c = 2; c <= num - 1; c++)
        { 
            if (num % c == 0) //not prime
            {
                break;
            }
        }

        if (c == num) //prime
        {
            int pos = atomicAdd(&device_primes[0],1);
            device_primes[pos] = num;

        }

        num += blockDim.x * gridDim.x; // Next number for this thread       
    }
}
