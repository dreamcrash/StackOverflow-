
#include <hip/hip_runtime.h>
/** Original Question : https://stackoverflow.com/questions/13301309/

I'm working on a statistical application containing approximately 10 - 30 million floating point values in an array.

Several methods performing different, but independent, calculations on the array in nested loops, for example:

Dictionary<float, int> noOfNumbers = new Dictionary<float, int>();

for (float x = 0f; x < 100f; x += 0.0001f) {
    int noOfOccurrences = 0;

    foreach (float y in largeFloatingPointArray) {
        if (x == y) {
            noOfOccurrences++;
        }
    }

    noOfNumbers.Add(x, noOfOccurrences);
}
The current application is written in C#, runs on an Intel CPU and needs several hours to complete. I have no knowledge of GPU programming concepts and APIs, so my questions are:

Is it possible (and does it make sense) to utilize a GPU to speed up such calculations?
If yes: Does anyone know any tutorial or got any sample code (programming language doesn't matter)?

**/

__global__ void hash (float *largeFloatingPointArray,int largeFloatingPointArraySize, int *dictionary, int size, int num_blocks)
{
    int x = (threadIdx.x + blockIdx.x * blockDim.x); // Each thread of each block will
    float y;                                         // compute one (or more) floats
    int noOfOccurrences = 0;
    int a;
    
    while( x < size )            // While there is work to do each thread will:
    {
        dictionary[x] = 0;       // Initialize the position in each it will work
        noOfOccurrences = 0;    

        for(int j = 0 ;j < largeFloatingPointArraySize; j ++) // Search for floats
        {                                                     // that are equal 
                                                             // to it assign float
           y = largeFloatingPointArray[j];  // Take a candidate from the floats array 
           y *= 10000;                      // e.g if y = 0.0001f;
           a = y + 0.5;                     // a = 1 + 0.5 = 1;
           if (a == x) noOfOccurrences++;    
        }                                      
                                                    
        dictionary[x] += noOfOccurrences; // Update in the dictionary 
                                          // the number of times that the float appears 

    x += blockDim.x * gridDim.x;  // Update the position here the thread will work
    }
}
